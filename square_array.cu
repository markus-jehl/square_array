#include "hip/hip_runtime.h"
#include "square_array.h"
#include "square_op.h"
#include "debug.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdexcept>

__global__ void square_kernel(float* data, size_t n, float* sum) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        compute_and_accumulate(data, idx, sum);
    }
}

void square_array(float* array, size_t size, float* result_sum, int device_id) {

    float* device_array = nullptr;
    float* device_sum = nullptr;
    bool needs_copy_back = false;
    float zero = 0.0f;

    hipPointerAttribute_t attr;
    hipError_t err = hipPointerGetAttributes(&attr, array);

    bool is_device_ptr = false;

//#if CUDART_VERSION >= 10000
    if (err == hipSuccess && (attr.type == hipMemoryTypeDevice || attr.type == hipMemoryTypeManaged)){
        is_device_ptr = true;
        DEBUG_PRINT("Array is on device: %d\n", attr.device);
    }
//#else
//    if (err == hipSuccess && attr.memoryType == hipMemoryTypeDevice)
//        is_device_ptr = true;
//#endif

    if (is_device_ptr) {
        device_array = array;
        // use the device ID of the pointer to set the device
        hipSetDevice(attr.device);
    } else {

        // set the device ID as specified
        int device_count = 0;
        hipGetDeviceCount(&device_count);
        if (device_id >= device_count) {
            throw std::runtime_error("Invalid CUDA device ID: " + std::to_string(device_id));
        }

        DEBUG_PRINT("Copying array from host to device\n");
        hipSetDevice(device_id);

        hipMalloc(&device_array, size * sizeof(float));
        hipMemcpy(device_array, array, size * sizeof(float), hipMemcpyHostToDevice);
        needs_copy_back = true;
    }

    // get and print the current cuda device ID
    int current_device_id;
    hipGetDevice(&current_device_id);
    DEBUG_PRINT("Using CUDA device: %d\n", current_device_id);

    // Allocate memory for the sum on the device
    hipMalloc(&device_sum, sizeof(float));
    hipMemcpy(device_sum, &zero, sizeof(float), hipMemcpyHostToDevice);

    int threads = 256;
    int blocks = (int)((size + threads - 1) / threads);
    square_kernel<<<blocks, threads>>>(device_array, size, device_sum);
    hipDeviceSynchronize();

    if (needs_copy_back) {
        hipMemcpy(array, device_array, size * sizeof(float), hipMemcpyDeviceToHost);
        hipFree(device_array);
    }

    hipMemcpy(result_sum, device_sum, sizeof(float), hipMemcpyDeviceToHost);
    hipFree(device_sum);
}

